#include "hip/hip_runtime.h"
// Program : rdma_gpu.cu
// Author : Sousuke Kanamoto

#include <hip/hip_runtime.h>
#include "rdma_common.h"

struct ibv_mr *mr = NULL;
int totalsize, buffsize;
void *addr;

/* This function outputs RDMA memory region (called from cpu) */
__global__ void output_host(char *array)
{
	printf("rdma_buffer: '%s'\n",array);
}

/* This function outputs RDMA memory region (called from gpu) */
__device__ void output_device(char *array)
{
	printf("rdma_buffer: '%s'\n",array);
}

/* This function is memcmp (for gpu) */
__device__ int memcompare(const void *s1, const void *s2, size_t n)
{
	register const unsigned char *ss1, *ss2, *t;
	int result = 0;

	for (ss1 = (const unsigned char *)s1, ss2 = (const unsigned char *)s2, t = ss2 + n;
    	ss2 != t && (result = *ss1 - *ss2) == 0;
    	ss1++, ss2++);

	return result;
}

/* This function registers RDMA memory region on GPU */
extern "C" struct ibv_mr* rdma_gpubuffer_alloc(struct ibv_pd *pd, uint32_t length,
    enum ibv_access_flags permission)
{
	if (!pd) {
		rdma_error("Protection domain is NULL \n");
		return NULL;
	}
	hipMalloc((void**)&addr, length);
	if (!addr) {
		rdma_error("failed to allocate buffer, -ENOMEM\n");
		return NULL;
	}
	debug("Buffer allocated: %p , len: %u \n", addr, length);

	if (!pd) {
		rdma_error("Protection domain is NULL, ignoring \n");
		return NULL;
	}
	mr = ibv_reg_mr(pd, addr, length, permission);
	if (!mr) {
		rdma_error("Failed to create mr on buffer, errno: %d \n", -errno);
		hipFree(addr);
	}
	debug("Registered: %p , len: %u , stag: 0x%x \n", 
	      mr->addr, 
	      (unsigned int) mr->length, 
	      mr->lkey);

	totalsize = length/sizeof(char); /* Total memory region size (buffsize*2) */
	buffsize = totalsize/2; /* Memory region size (RDMA Write or RDMA Read) */

	return mr;
}

/* This function releases RDMA memory region on GPU */
extern "C" void rdma_gpubuffer_free()
{
        if (!mr) {
	        rdma_error("Passed memory region is NULL, ignoring\n");
		return ;
	}
	void *to_free = mr->addr;

	debug("Deregistered: %p , len: %u , stag : 0x%x \n", 
	      mr->addr, 
	      (unsigned int) mr->length, 
	      mr->lkey);
	ibv_dereg_mr(mr);

	debug("Buffer %p free'ed\n", to_free);
	hipFree(to_free);
}

/* This function is kernel on GPU */
__global__ void kernel(void *addr, int totalsize, int buffsize)
{
        char src[] = "input", *s = src;
	char dst[] = "output", *d = dst;

	/* Wait RDMA Write and write strings */
	while(1){
	        if(memcompare((void *)addr, (void *)s, 5)==0){ /* Wait RDMA Write */
		        memcpy((void *)addr+buffsize, (const void *)d, 6); /* Write data to RDMA buffer */
			break;
		}
	}
}

/* This function starts kernel on GPU */
extern "C" void kernel_start()
{	
	kernel<<<1, 1>>>(addr, totalsize, buffsize);
}
